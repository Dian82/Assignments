#include "hip/hip_runtime.h"
/*
 * This code is part of the hw2 of multicore programming in SYSU
 * Copyright (c) 2020 Hongzheng Chen
 * Email: chenhzh37@mail2.sysu.edu.cn
 * 
 * This file is the kernel part of CUDA implementation
 * of calculating nearest neighbor in high dimension.
 *
 * This program is a baseline implementation.
 */

#include "core.h"
#define MAX_Q 1024

/*!
 * Naive CPU implementation
 *   used to test the correctness of the results
 * \param k The dimension size of the points
 * \param m The nubmer of search points
 * \param n The number of reference points
 * \param searchPoints
 * \param referencePoints
 * \param results
 * \return void. Results will be put in result.
 */
extern void cudaCallbackCPU(int k, int m, int n, float *searchPoints,
                            float *referencePoints, int **results) {

    int *tmp = (int*)malloc(sizeof(int)*m);
    int minIndex;
    float minSquareSum, diff, squareSum;

    // Iterate over all search points
    for (int mInd = 0; mInd < m; mInd++) {
        minSquareSum = -1;
        // Iterate over all reference points
        for (int nInd = 0; nInd < n; nInd++) {
            squareSum = 0;
            for (int kInd = 0; kInd < k; kInd++) {
                diff = searchPoints[k*mInd+kInd] - referencePoints[k*nInd+kInd];
                squareSum += (diff * diff);
            }
            if (minSquareSum < 0 || squareSum < minSquareSum) {
                minSquareSum = squareSum;
                minIndex = nInd;
            }
        }
        tmp[mInd] = minIndex;
    }

    *results = tmp;
    // Note that you don't have to free searchPoints, referencePoints, and
    // *results by yourself
}

/*!
 * Core execution part of CUDA
 *   that calculates the nearest neighbor of each search point.
 * \param k The dimension size of the points
 * \param m The nubmer of search points
 * \param n The number of reference points
 * \param searchPoints
 * \param referencePoints
 * \param output
 * \return void. Results will be put in output
 */
__global__ void kernel(int k, int m, int n, float* searchPoints, float* referencePoints, int* output) {
    // int pid = threadIdx.x;
    int pid = blockIdx.x * blockDim.x + threadIdx.x;
    int minIdx;
    float minSquareSum = -1;
    float diff, squareSum;
    // Iterate over all reference points
    if (pid < m) {
        for (int nInd = 0; nInd < n; nInd++) { // ref points
            squareSum = 0;
            for (int kInd = 0; kInd < k; kInd++) { // dimension
                diff = searchPoints[k * pid + kInd]
                     - referencePoints[k * nInd + kInd];
                squareSum += (diff * diff);
            }
            if (minSquareSum < 0 || squareSum < minSquareSum) {
                minSquareSum = squareSum;
                minIdx = nInd;
            }
        }
        output[pid] = minIdx;
    }
}

/*!
 * Wrapper of the CUDA kernel
 *   used to be called in the main function
 * \param k The dimension size of the points
 * \param m The nubmer of search points
 * \param n The number of reference points
 * \param searchPoints
 * \param referencePoints
 * \param results
 * \return void. Results will be put in result.
 */
extern void cudaCallbackGPU_baseline(int k, int m, int n, float *searchPoints,
                                     float *referencePoints, int **results) {
    float *searchPoints_d, *referencePoints_d;
    int* output_d;

    // Allocate device memory and copy data from host to device
    CHECK(hipMalloc((void **)&searchPoints_d, sizeof(float)*m*k));
    CHECK(hipMalloc((void **)&referencePoints_d, sizeof(float)*n*k));
    CHECK(hipMalloc((void **)&output_d, sizeof(int)*m));
    CHECK(hipMemcpy(searchPoints_d, searchPoints, sizeof(float)*m*k, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(referencePoints_d, referencePoints, sizeof(float)*n*k, hipMemcpyHostToDevice));

    // Invoke the device function
    kernel<<< 1,m >>>(k, m, n, searchPoints_d, referencePoints_d, output_d);
    hipDeviceSynchronize();

    // Copy back the results and de-allocate the device memory
    *results = (int *)malloc(sizeof(int)*m);
    assert(results != NULL);
    CHECK(hipMemcpy(*results, output_d, sizeof(int)*m, hipMemcpyDeviceToHost));

    // int *cpu_results;
    // cudaCallbackCPU(k, m, n, searchPoints, referencePoints, &cpu_results);
    // for (int i = 0; i < m; ++i)
    //     assert(cpu_results[i] == (*results)[i]);

    CHECK(hipFree(searchPoints_d));
    CHECK(hipFree(referencePoints_d));
    CHECK(hipFree(output_d));
}

/*!
 * Core execution part of CUDA
 *   that calculates the nearest neighbor of each search point.
 * \param k The dimension size of the points
 * \param m The nubmer of search points
 * \param n The number of reference points
 * \param searchPoints
 * \param referencePoints
 * \param output
 * \return void. Results will be put in output
 */
__global__ void kernel_sharedmem(int k, int m, int n, float* searchPoints, float* referencePoints, int* output) {
    int bid = blockIdx.x; // each block, one search point
    int tid = threadIdx.x;
    int n_points = ((n % 1024 == 0) ? n / 1024 : n / 1024 + 1);
    int searchId = bid;
    int referenceId = tid; // [tid,tid+n_points]
    int minIdx;
    float diff, squareSum;
    __shared__ float s_mem[16];
    if (tid < k) {
        s_mem[tid] = searchPoints[k * searchId + referenceId];
    }
    __syncthreads();
    __shared__ float dist[1024];
    __shared__ int dist_idx[1024];
    float minSquareSum = -1;
    for (int i = 0; i < n_points; ++i) {
        squareSum = 0;
        int refId = referenceId * n_points + i;
        for (int kInd = 0; kInd < k; kInd++) { // dimension
            diff = s_mem[kInd] - referencePoints[k * refId + kInd];
            squareSum += (diff * diff);
        }
        if (minSquareSum < 0 || squareSum < minSquareSum) {
            minSquareSum = squareSum;
            minIdx = refId;
        }
    }
    dist[referenceId] = minSquareSum;
    dist_idx[referenceId] = minIdx;
    __syncthreads();
    if (referenceId == 0) {
        float minSquareSum = -1;
        for (int i = 0; i < 1024; ++i) {
            squareSum = dist[i];
            if (minSquareSum < 0 || squareSum < minSquareSum) {
                minSquareSum = squareSum;
                minIdx = dist_idx[i];
            }
        }
        output[searchId] = minIdx;
    }
}

/*!
 * Wrapper of the CUDA kernel
 *   used to be called in the main function
 * \param k The dimension size of the points
 * \param m The nubmer of search points
 * \param n The number of reference points
 * \param searchPoints
 * \param referencePoints
 * \param results
 * \return void. Results will be put in result.
 */
extern void cudaCallbackGPU_sharedmem(int k, int m, int n, float *searchPoints,
                                      float *referencePoints, int **results) {
    float *searchPoints_d, *referencePoints_d;
    int* output_d;

    // Allocate device memory and copy data from host to device
    CHECK(hipMalloc((void **)&searchPoints_d, sizeof(float)*m*k));
    CHECK(hipMalloc((void **)&referencePoints_d, sizeof(float)*n*k));
    CHECK(hipMalloc((void **)&output_d, sizeof(int)*m));
    CHECK(hipMemcpy(searchPoints_d, searchPoints, sizeof(float)*m*k, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(referencePoints_d, referencePoints, sizeof(float)*n*k, hipMemcpyHostToDevice));

    // Invoke the device function
    kernel_sharedmem<<< m, 1024 >>>(k, m, n, searchPoints_d, referencePoints_d, output_d);
    hipDeviceSynchronize();

    // Copy back the results and de-allocate the device memory
    *results = (int *)malloc(sizeof(int)*m);
    assert(results != NULL);
    CHECK(hipMemcpy(*results, output_d, sizeof(int)*m, hipMemcpyDeviceToHost));

    // int *cpu_results;
    // cudaCallbackCPU(k, m, n, searchPoints, referencePoints, &cpu_results);
    // for (int i = 0; i < m; ++i)
    //     assert(cpu_results[i] == (*results)[i]);

    CHECK(hipFree(searchPoints_d));
    CHECK(hipFree(referencePoints_d));
    CHECK(hipFree(output_d));
}