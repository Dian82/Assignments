#include "hip/hip_runtime.h"
/*
 * This code is part of the hw1 of multicore programming in SYSU
 * Copyright (c) 2020 Hongzheng Chen
 * Email: chenhzh37@mail2.sysu.edu.cn
 * 
 * This file is the kernel part of CUDA implementation
 * that calculates the central entropy of each point in a matrix.
 *
 * This program is an optimized implementation using shared memory.
 */

#include "core.h"

#define blockW 16
#define blockH 16
#define RADIUS 2

/*!
 * Core execution part of CUDA
 *   that calculates the central entropy of each point.
 * \param size The size of the input matrix.
 * \param width The width of the input matrix.
 * \param height The height of the input matrix.
 * \param input The input matrix.
 * \param output The output matrix.
 * \return void. Results will be put in output.
 */
__global__ void kernel(int size, int width, int height, float *input, float *output) {
    // true index (x,y)
    const int x = blockIdx.x * blockW + threadIdx.x;
    const int y = blockIdx.y * blockH + threadIdx.y;
    const int idx = y * width + x;
    // only those threads in the window need to be calculated
    if (x >= 0 && x < width && y >= 0 && y < height) {
        // each thread first counts the histogram of idx
        int cnt[16] = {0}; // histogram
        int valid = 0;
        for (int i = -2; i < 3; ++i)
            for (int j = -2; j < 3; ++j) {
                if (y + i >= 0 && y + i < height &&
                    x + j >= 0 && x + j < width) {
                    int in = input[idx + i * width + j];
                    cnt[in]++;
                    valid++;
                }
            }
        // calculate entropy
        float sum = 0;
        for (int i = 0; i < 16; ++i) {
            int ni = cnt[i];
            if (ni != 0) {
                #ifdef LOOKUP
                sum += ni * log_table[ni];
                #else
                sum += ni * logf(ni);
                #endif
            }
        }
        #ifdef LOOKUP
        output[idx] = -sum / valid + log_table[valid];
        #else
        output[idx] = -sum / valid + logf(valid);
        #endif
    }
}

/*!
 * Wrapper of the CUDA kernel
 *   used to be called in the main function
 * \param width The width of the input matrix.
 * \param height The height of the input matrix.
 * \param sample The input matrix.
 * \param result The output matrix.
 * \return void. Results will be put in result.
 */
void cudaCallback(int width, int height, float *sample, float **result) {
    int size = width * height;
    float *input_d, *output_d;

    // Allocate device memory and copy data from host to device
    CHECK(hipMalloc((void **)&input_d, sizeof(float)*size));
    CHECK(hipMalloc((void **)&output_d, sizeof(float)*size));
    CHECK(hipMemcpy(input_d, sample, sizeof(float)*size, hipMemcpyHostToDevice));

    printf("grid: %d %d\n",divup(width, blockW),divup(height, blockH));
    printf("block size: %d %d\n",blockW,blockH);
    // Invoke the device function
    const dim3 grid(divup(width, blockW), divup(height, blockH));
    const dim3 threadBlock(blockW, blockH);
    kernel<<< grid, threadBlock >>>(size, width, height, input_d, output_d);
    hipDeviceSynchronize();

    // Copy back the results and de-allocate the device memory
    *result = (float *)malloc(sizeof(float)*size);
    CHECK(hipMemcpy(*result, output_d, sizeof(float)*size, hipMemcpyDeviceToHost));
    CHECK(hipFree(input_d));
    CHECK(hipFree(output_d));

    // Note that you don't have to free sample and *result by yourself
}